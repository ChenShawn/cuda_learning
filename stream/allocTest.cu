
#include <hip/hip_runtime.h>
#include <cstdio>
template<typename T>
__device__ __inline__ void add(T& val) {
	val += 1;
}

template<typename T>
__global__ void func(T* ptr) {
	add<T>(ptr[blockIdx.x]);
}

int main() {
	hipStream_t stream;
	hipStreamCreate(&stream);
	int *h_ptr, *d_ptr;
	hipHostAlloc(&h_ptr, 20, hipHostMallocDefault);
	for(int i=0; i<5; i++) {
		h_ptr[i] = i*2 + 1;
	}
	hipMalloc((void**)&d_ptr, 20);
	hipMemcpyAsync(d_ptr, h_ptr, 20, hipMemcpyHostToDevice, stream);
	func<<<5, 1, 0, stream>>>(d_ptr);
	hipMemcpyAsync(h_ptr, d_ptr, 20, hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
	for(int i=0; i<5; i++) {
		printf("%d ", h_ptr[i]);
	}
	printf("\n");
	return 0;
}


